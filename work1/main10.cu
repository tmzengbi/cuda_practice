#include <hip/hip_runtime.h>
#include <cstdio>

const int N = 1 << 14;

/**
 * @brief using global memory
 */
__global__ void calc(/*float *A, float *x, */float *y) {
    // normal implementation
    // for(int i = 0; i < N; ++ i)
    //     for(int j = 0; j < N; ++ j)
    //         y[i] += A[i][j] * x[j];
    // printf("%d %d %d %d\n",blockDim.x, blockIdx.x, gridDim.x, threadIdx.x);
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int step = blockDim.x * gridDim.x;
#define A(i,j) (i - 0.1 * j + 1)
#define x(i) log(sqrt(i * i - i + 2.0));
    for(int i = tid; i < N; i += step)
        for(int j = 0; j < N; ++ j)
            // y[i] += A[idA] * x[j];
            y[i] += A(i,j) * x(j);
#undef A
#undef x
}

#ifdef NDEBUG
    #define cudaWork(work) work
#else
    #define cudaWork(work) \
        do { \
            work; \
            hipError_t err = hipGetLastError(); \
            if(err) { \
                printf("error occur in line %d, %s\n", __LINE__, hipGetErrorString(err)); \
                exit(EXIT_FAILURE); \
            } \
        } while(0)
#endif
int main() {
    float *y, *res;
    res = (float*) malloc(sizeof(float) * N);
    cudaWork(hipMalloc(&y, sizeof(float) * N));
    hipEvent_t start, stop;
    float t = 0;
    cudaWork(hipEventCreate(&start));
    cudaWork(hipEventCreate(&stop));
    cudaWork(hipEventRecord(start));
    cudaWork((calc<<<2,16>>>(y)));
    cudaWork(hipEventRecord(stop));
    cudaWork(hipEventSynchronize(stop));
    cudaWork(hipEventElapsedTime(&t, start, stop));
    printf("elapsedTime %fms\n", t);
    cudaWork(hipMemcpy(res, y, sizeof(float) * N, hipMemcpyDeviceToHost));
    // for(int i = 0; i < N ; ++ i)
    //     printf("%f\n", res[i]);
    
    hipFree(y);
    free(res);
    
}
