#include <hip/hip_runtime.h>
#include <cstdio>

const int N = 1 << 14;
const int threadTot = 1024;

/**
 * @brief using global memory
 */
__global__ void calc(/*double *A, double *x, */double *y) {
    // normal implementation
    // for(int i = 0; i < N; ++ i)
    //     for(int j = 0; j < N; ++ j)
    //         y[i] += A[i][j] * x[j];
#define A(i,j) (i - 0.1 * j + 1)
#define x(i) logf(sqrtf(i * i - i + 2.0))
    for(int i = threadIdx.x; i < N; i += blockDim.x)
        for(int j = 0; j < N; ++ j)
            // y[i] += A[idA] * x[j];
            y[i] += A(i,j) * x(j);
#undef A
#undef x
}

#ifdef NDEBUG
    #define cudaWork(work) work
#else
    #define cudaWork(work) \
        do { \
            work; \
            hipError_t err = hipGetLastError(); \
            if(err) { \
                printf("error occur in line %d, %s\n", __LINE__, hipGetErrorString(err)); \
                exit(EXIT_FAILURE); \
            } \
        } while(0)
#endif
int main() {
    double *y, *res;
    res = (double*) malloc(sizeof(double) * N);
    cudaWork(hipMalloc(&y, sizeof(double) * N));
    hipEvent_t start, stop;
    float t = 0;
    cudaWork(hipEventCreate(&start));
    cudaWork(hipEventCreate(&stop));
    cudaWork(hipEventRecord(start));
    calc<<<1,threadTot>>>(y);cudaWork();
    cudaWork(hipEventRecord(stop));
    cudaWork(hipEventSynchronize(stop));
    cudaWork(hipEventElapsedTime(&t, start, stop));
    printf("elapsedTime %fms\n", t);
    cudaWork(hipMemcpy(res, y, sizeof(double) * N, hipMemcpyDeviceToHost));
    // for(int i = 0; i < N ; ++ i)
    //     printf("%f\n", res[i]);
    
    hipFree(y);
    free(res);
}
