#include <hip/hip_runtime.h>
#include <cstdio>

const int N = 1 << 14;

/**
 * @brief constant memory
 * 
 */ 
__constant__ float x_constDevice[N];
__global__ void calc(/*float *A, float *x,*/ float *y) {
    // normal implementation
    // for(int i = 0; i < N; ++ i)
    //     for(int j = 0; j < N; ++ j)
    //         y[i] += A[i][j] * x[j];
    // printf("%d %d %d %d\n",blockDim.x, blockIdx.x, gridDim.x, threadIdx.x);
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int step = blockDim.x * gridDim.x;
#define A(i,j) (i - 0.1 * j + 1)
    for(int i = tid; i < N; i += step)
        for(int j = 0; j < N; ++ j)
            // y[i] += A[idA] * x[j];
            y[i] += A(i,j) * x_constDevice[j];
#undef A
}
#ifdef NDEBUG
    #define cudaWork(work) work
#else
    #define cudaWork(work) \
        do { \
            work; \
            hipError_t err = hipGetLastError(); \
            if(err) { \
                fprintf(stderr, "error occur in line %d, %s\n", __LINE__, hipGetErrorString(err)); \
                exit(EXIT_FAILURE); \
            } \
        } while(0)
#endif

int main() {
    float *x_host, *y, *res;
    x_host = (float*) malloc(sizeof(float) * N);
    res = (float*) malloc(sizeof(float) * N);
    cudaWork(hipMalloc(&y, sizeof(float) * N));
    for(int i = 0; i < N; ++ i)
        x_host[i] = log(sqrt(i * i - i + 2.0));
    cudaWork(hipMemcpyToSymbol(HIP_SYMBOL(x_constDevice), x_host, sizeof(float) * N));
    hipEvent_t start, stop;
    float t = 0;
    cudaWork(hipEventCreate(&start));
    cudaWork(hipEventCreate(&stop));
    cudaWork(hipEventRecord(start));
    cudaWork((calc<<<2,16>>>(y)));
    cudaWork(hipEventRecord(stop));
    cudaWork(hipEventSynchronize(stop));
    cudaWork(hipEventElapsedTime(&t, start, stop));
    printf("elapsedTime %fms\n", t);
    cudaWork(hipMemcpy(res, y, sizeof(float) * N, hipMemcpyDeviceToHost));
    // for(int i = 0; i < N ; ++ i)
    //     printf("%f\n", res[i]);
    
    hipFree(y);
    free(res);
    free(x_host);
    
}
