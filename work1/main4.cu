#include <hip/hip_runtime.h>
#include <cstdio>

const int N = 1 << 14;
/**
 * @brief shared memory
 * 
 */ 
extern __shared__ float x_shared[];
__global__ void calc(/*float *A, float *x,*/ float *y) {
    // normal implementation
    // for(int i = 0; i < N; ++ i)
    //     for(int j = 0; j < N; ++ j)
    //         y[i] += A[i][j] * x[j];
    // printf("%d %d %d %d\n",blockDim.x, blockIdx.x, gridDim.x, threadIdx.x);
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int step = blockDim.x * gridDim.x;
    for(int i = tid; i < N; i += step)
        x_shared[i] = log(sqrt(i * i - i + 2.0));
    __syncthreads();
#define A(i,j) (i - 0.1 * j + 1)
    for(int i = tid; i < N; i += step)
        for(int j = 0; j < N; ++ j)
            // y[i] += A[idA] * x[j];
            y[i] += A(i,j) * x_shared[j];
#undef A
}
#ifdef NDEBUG
    #define cudaWork(work) work
#else
    #define cudaWork(work) \
        do { \
            work; \
            hipError_t err = hipGetLastError(); \
            if(err) { \
                fprintf(stderr, "error occur in line %d, %s\n", __LINE__, hipGetErrorString(err)); \
                exit(EXIT_FAILURE); \
            } \
        } while(0)
#endif
int main() {
    float *y, *res;
    res = (float*) malloc(sizeof(float) * N);
    cudaWork(hipMalloc(&y, sizeof(float) * N));
    hipEvent_t start, stop;
    float t = 0;
    cudaWork(hipEventCreate(&start));
    cudaWork(hipEventCreate(&stop));
    cudaWork(hipEventRecord(start));
    hipFuncSetAttribute(reinterpret_cast<const void*>(calc), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
    cudaWork((calc<<<2, 16, N * sizeof(float)>>>(y)));
    cudaWork(hipEventRecord(stop));
    cudaWork(hipEventSynchronize(stop));
    cudaWork(hipEventElapsedTime(&t, start, stop));
    printf("elapsedTime %fms\n", t);
    cudaWork(hipMemcpy(res, y, sizeof(float) * N, hipMemcpyDeviceToHost));
    // for(int i = 0; i < N ; ++ i)
    //     printf("%f\n", res[i]);
    
    hipFree(y);
    free(res);
    
}
