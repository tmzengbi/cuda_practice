#include <hip/hip_runtime.h>
#include <cstdio>

const int N = 1 << 14;
const int Dim = 32;

/**
 * @brief coalesced memory
 * no need for the optimization
 */ 

__global__ void calc();

#ifdef NDEBUG
    #define cudaWork(work) work
#else
    #define cudaWork(work) \
        do { \
            work; \
            hipError_t err = hipGetLastError(); \
            if(err) { \
                printf("error occur in line %d, %s\n", __LINE__, hipGetErrorString(err)); \
                exit(EXIT_FAILURE); \
            } \
        } while(0)
#endif
int main() {
    
}
