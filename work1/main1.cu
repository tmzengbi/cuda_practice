#include <hip/hip_runtime.h>
#include <cstdio>

const int N = 1 << 14;

/**
 * @brief using global memory
 */
__global__ void calc(/*float *A,*/ float *x, float *y) {
    // normal implementation
    // for(int i = 0; i < N; ++ i)
    //     for(int j = 0; j < N; ++ j)
    //         y[i] += A[i][j] * x[j];
    // printf("%d %d %d %d\n",blockDim.x, blockIdx.x, gridDim.x, threadIdx.x);
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int step = blockDim.x * gridDim.x;
#define A(i,j) (i - 0.1 * j + 1)
    for(int i = tid; i < N; i += step)
        for(int j = 0; j < N; ++ j)
            // y[i] += A[idA] * x[j];
            y[i] += A(i,j) * x[j];
#undef A
}

#ifdef NDEBUG
    #define cudaWork(work) work
#else
    #define cudaWork(work) \
        do { \
            work; \
            hipError_t err = hipGetLastError(); \
            if(err) { \
                printf("error occur in line %d, %s\n", __LINE__, hipGetErrorString(err)); \
                exit(EXIT_FAILURE); \
            } \
        } while(0)
#endif
int main() {
    float *x_host, *x, *y, *res;
    x_host = (float*) malloc(sizeof(float) * N);
    cudaWork(hipMalloc(&x, sizeof(float) * N));
    res = (float*) malloc(sizeof(float) * N);
    cudaWork(hipMalloc(&y, sizeof(float) * N));
    for(int i = 0; i < N; ++ i)
        x_host[i] = log(sqrt(i * i - i + float(2.0)));
    cudaWork(hipMemcpy(x, x_host, sizeof(float) * N, hipMemcpyHostToDevice));
    hipEvent_t start, stop;
    float t = 0;
    cudaWork(hipEventCreate(&start));
    cudaWork(hipEventCreate(&stop));
    cudaWork(hipEventRecord(start));
    cudaWork((calc<<<2,16>>>(x, y)));
    cudaWork(hipEventRecord(stop));
    cudaWork(hipEventSynchronize(stop));
    cudaWork(hipEventElapsedTime(&t, start, stop));
    printf("elapsedTime %fms\n", t);
    cudaWork(hipMemcpy(res, y, sizeof(float) * N, hipMemcpyDeviceToHost));
    // for(int i = 0; i < N ; ++ i)
    //     printf("%f\n", res[i]);
    
    hipFree(y);
    hipFree(x);
    free(res);
    free(x_host);
    
}
